#include "hip/hip_runtime.h"
/*
 * BigInteger.c
 *
 *  Created on: 18 ene. 2019
 *    Author: DoHITB under MIT License
 *
 * +--------------------------------------------------------------------------------+
 * | MIT License                                                                    |
 * |                                                                                |
 * | Copyright (c) 2022 David Oscar Solé González (aka DoHITB)                      |
 * |                                                                                |
 * | Permission is hereby granted, free of charge, to any person obtaining a copy   |
 * | of this software and associated documentation files (the "Software"), to deal  |
 * | in the Software without restriction, including without limitation the rights   |
 * | to use, copy, modify, merge, publish, distribute, sublicense, and/or sell      |
 * | copies of the Software, and to permit persons to whom the Software is          |
 * | furnished to do so, subject to the following conditions:                       |
 * |                                                                                |
 * | The above copyright notice and this permission notice shall be included in all |
 * | copies or substantial portions of the Software.                                |
 * |                                                                                |
 * | THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR     |
 * | IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,       |
 * | FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE    |
 * | AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER         |
 * | LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,  |
 * | OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE  |
 * | SOFTWARE.                                                                      |
 * +--------------------------------------------------------------------------------+
 *
 *  CHANGELOG
 *  v1.1 (beta)
 *    - Review variables and functions, making it "static" when convenient.
 *    - Changing all pointer references to "void" to avoid memory leakage.
 *    - Added new validations on BI creation.
 *    - New function "BImemcpy" to retreive useful values of BigInteger.
 *    - New function "validateBI" to validate the internal structure of a BI void pointer.
 *  v1.11 (beta)
 *    - Added memory cleansing before "return" statements.
 *  v1.2 (beta)
 *    - Added new behaviour for better performance on "pow" function.
 *  v1.3 (beta)
 *    - Patch to avoid memory leackage.
 *    - Deleted "time.h" library (unnecesary).
 *  v1.4 (beta)
 *    - Changed "BIQSRT" function for "nqrt", that makes "n-root"
 *    - Delete "static" from BIT (unnecesary).
 *    - Created wrapper function for "add" ,"sub", and "mul" for internal use.
 *    - Arranged header variable names to naming convention.
 *    - Function "signum" modified to increase performance. It now works with two int.
 *      - 90,566% memory usage reduction on nqrt via wrappers.
 *      - 53,656% performance increase on nqrt via wrappers.
 *      - (nqrt is used to measurement as nqrt -> pow -> mul -> pmul -> add is the longest function chain available).
 *  v1.5 (beta)
 *    - Changes on toString function to move data to char  instead of printing.
 *  v1.6 (beta)
 *    - Bugfix on validateBI on negative numbers.
 *    - Bugfix on toString on negative numbers.
 *  v2.0 (Release)
 *    - SIT (324 diferent operations) done. Fixes done.
 *      - Remove validateBI from pSub.
 *      - Bugfix on add if b < 0 and |a| < |b|.
 *      - Bugfix on add if |a| < |b| and a, b < 0.
 *      - Bugfix on division if a / b, b < 0.
 *    - Code cleansing
 *  v2.1 (Release)
 *    - Bugfix on "division" function.
 *  v2.2 (Release)
 *    - Bugfix on "validateBI" function. Avoid "t" data leackage.
 *  v2.3 (Release)
 *    - Naming change on "pow" function. Now it's called "bipow" to avoid internal errors.
 *  v2.4 (Release)
 *    - Bugfix on division when len(a) - len(b) = 1.
 *  v2.5 (Release)
 *    - Change on "nqrt" function for performance improvement. Root calculation via Bolzano.
 *  v2.51 (Release)
 *    - Bugfix on "validateBI".
 *  v3.0 (Major Release)
 *    - Deleted unnecesary commented lines.
 *    - Changed the signature of "subtraction" for "subtract(a, b)".
 *    - Deleted function "rev" (unnecesary).
 *    - Deleted "min" variable from multiplication.
 *    - Deleted "a = -1" and "b = -1" validation on multiplication (unnecesary).
 *    - Added length on "pMul" function to avoid heap overflow.
 *    - Change on "sMul" function. Instead of doing all the products and then all the add, now it make product-addition loop. This way, there are less memory consumption and better performance.
 *    - Change on "carryAdd" function. Deleted "Do/While" loop (unncesary).
 *    - Deleted function "makeBase" (unnecesary).
 *    - Deleted function "shift" (unnecesary).
 *    - New function iniStr. It allocate memory for "toString" function.
 *    - General code cleansing.
 *  v3.01 (Release)
 *    - Deleted "Do/While" loop on "carrySub" function.
 *  v3.02 (Release)
 *    - Optimization on "carryAdd" function. Carry is made in two times.
 *      - Common part: it's all carried.
 *      - Non-common part: it's carried unti acc = 0.
 *      - 3.448% performance increase tested with 10.000 additions of 100 digit numbers.
 *  v3.1 (Release)
 *    - Function "nqrt" modified.
 *      - Change on "add" calls by "pAdd".
 *      - Change on "equals" calls by "hardEquals".
 *      - Change on "sub" calls by "pSub".
 *      - Change on "bipow" calls by "sBipow".
 *      - Performance increase of 38'758% based on 10.000 nqrt calls.
 *    - New function "sBipow", static wrapper for "bipow".
 *    - Bugfix on "carryAdd".
 *  v3.11 (Release)
 *    - Deleted unused variables.
 *  v4.0 (Major Release)
 *    - Review for CUDA codification.
 *      - Changes on all functionalities. We operate directly upon the pointer, saving a bunch of performance (e.g: up to 25'957% on addition operation). Also reduce RAM consumption to 50%.
 *      - Additional validations on "nqrt" function.
 *      - General code cleansing.
 *      - Review and removal of "showError" codes.
 *      - Overall, 201 code lines were removed on the optimisation.
 *    - Bugfix on "validateBI".
 *    - Changes on "dvs" to delegate on static function "sDvs".
 *    - Changes on "nqrt" to delegate on static funcion "sNqrt".
 *    - New function "free" to massive memory free.
 *    - Bugfix on "toString" with negative numbers.
 *    - Some variables are changed to static.
 *    - Deleted "limits.h" library.
 *    - Component Test performed. Changes derived of it:
 *      - Added default value on "BImemcpy": 0.
 *      - Added a "clean" call on pAppend.
 *      - Bugfix on "append" with negative numbers.
 *      - Bugfix on division (0 / 0 = 1).
 *      - Bugifx on 0 or below index root.
 *      - Bugfix on below 0 index power.
 *  v4.1 (Release)
 *    - Bugfix on subtraction when a < b.
 *    - Added cast to int on strlen.
 *  v4.2 (Release)
 *    - Bugfix on "b" when a < b.
 *  v4.3 (Release)
 *    - Bugfix on one-digit division.
 *    - Bugfix on bipow when va < 0.
 *  v4.4 (Release)
 *    - New static BI data.
 *    - Improvement on BI_initialize.
 *    - Bugfix on same-operator functions.
 *      - add(a, a).
 *      - sub(a, a).
 *      - mul(a, a).
 *      - dvs(a, a).
 *  v4.5 (Release)
 *    - carryAdd is no longer static.
 *  v4.6 (Release)
 *    - add new field "k" on BigInteger, as preparation for Major Release BigInteger v5.0.
 *  v4.7 (Release)
 *    - "validateBI" changes to improve performance.
 *    - Deleted cast on malloc to imporve performance.
 *    - Bugfix on sNqrt when data lenght < 0.
 *    - Performance improvement on clean function.
 *  v4.71 (Release)
 *    - Added function iniBIT to allocate BIT memory.
 *    - Now multiplication and division works with external BIT to increase performance.
 *  v5.0 (Major Release)
 *    - Changed "n" type from int to char, reducing up to 75% of memory usage.
 *    - Use of external variables to increase performance (gaining memory usage).
 *    - "free" function has been removed.
 *    - New precompiler parameter added for setting BI length (C_MAX_LENGTH). If not present, 4096 will be set as length.
 *    - New precompiler parameter added for making BI validations optional (CVALIDATE). If it have a value of 1, validations will be made.
 *    - New precompiler parameter added or Standalone BI (BI_STANDALONE).
 *    - "iniBIT" function has been removed, as it has been integrated on "init" function.
 *    - Comment and identation revision.
 *    - Typedef created in order to remove "struct" references (code clarify).
 *    - Improvement made on "add" in order to improve performance.
 *    - Improvement made on "subtract". Now we call "addition" instead of "pAdd". Performance improvement.
 *    - Improvement made on "subtract". "CarryType" has been removed.
 *    - Improvement made on "mul". Now we call "addition" instead of "pAdd". Performance improvement.
 *    - Performance improvement made on "division" as "append" functionality has been rewritten.
 *       - "dtmp" variable removed.
 *       - "pAppend" is now an internal (static) function for "division" exclusive use.
 *    - Improvement made on "nqrt". Now we call "addition" instead of "pAdd". Performance improvement.
 *    - Performance improvement on "hardEquals" when "va" and "vb" are the same memory pointer.
 *    - "Clean" function is now public.
 *    - "BOperation" interface now created. Shall be used when BI_STANDALONE has not been declared.
 *      - Common functions have been refactorized.
 *      - _BI_Initialize not longer static.
 *    - New interface BigDouble created. Now you can use Integer and Double operations
 *      - "nqrt" and "bipow" are still integer-only operations.
 *  v5.01 (Release)
 *    - Bugfix on hardEquals function
 *  v5.02 (Release)
 *    - Improvement on "n" variable. Now it's signed char
 *  v5.1 (Release)
 *    - Included service mode by using precompiler directive "BI_SERVICE". If "BI_SERVICE == 1", program won't stop if error found.
 *    - Improved dynamically BigDouble creation by using precompiler directives
 *    - Bugfixes
 *  v5.2 (Release)
 *    - Bugfix on division while "b" have a signle digit
 *  v5.22 (Bugfix)
 *    - General bugfix on division
 *  v5.3 (Release)
 *    - Added new "mod" function that performs a %= b
 *  v5.31 (Bugfix)
 *    - Bugfix on "getMemorySize" function
 *  v5.32 (Bugfix)
 *    - Bugfix (again) on division function
 *  v6.0
 *    - CUDA Integration
 *      - Added precompiler directive "CUDA_ENABLED". While it's set to one, CUDA can be used.
 *      - "CUDA_ENABLED" directive shall be used aside with "BI_STANDALONE".
 *    - CUDA Functionalities
 *    - Comment translation begins
 *    - Function visibility (static) had been reviewed and improved by using "BI_STANDALONE" directive
 *    - Changes on mod signature to match the rest of the function signatures
 *    - Added validation on mod function
 *    - Removed "m" parameter for "equals" function
 *    - newBI function now has a "const" on chat to avoid compiling warnings
 *    - Changes on pSub to avoid stack loop issue during CUDA linking
 */

#include "string.h"
#include "stdio.h"
#include "stdlib.h"
#include "BigInteger.h"
#if BI_STANDALONE != 1
#include "BOperation.h"
#endif

#if CUDA_ENABLED == 1
#include "stdlib.h"
#include "conio.h"
#include "hip/hip_runtime.h"
#include ""
#endif

static float BI_VERSION = 6.0f;

#if BI_STANDALONE == 1
static int validate =
#ifdef CVALIDATE
CVALIDATE;
#else
1;
#endif
#endif

#if CUDA_ENABLED == 1
/*
 * h2d
 *
 * Copies host memory to device memory.
 * It assumes that both device and host are an array of quantity items.
 */
void h2d(void** device, void* host, int quantity, size_t size) {
  if (quantity <= 0) {
    showError(52);

    return;
  }

  hipMalloc(device, size * quantity);
  hipMemcpy(*device, host, size * quantity, hipMemcpyHostToDevice);
}

/*
 * d2h
 *
 * Copies device memory to host memory.
 * It assumes that both device and host are an array of quantity items.
 */
void d2h(void* host, void* device, int quantity, size_t size) {
  if (quantity <= 0) {
    showError(53);

    return;
  }

  hipMemcpy(host, device, size * quantity, hipMemcpyDeviceToHost);
}

/*
 * CUnewBI.
 *
 * Generates a new BI from the input string (dst).
 * The load is made on reverse order to allow a simple growing mechanism.
 * If "sig" is -1, the number will be negative.
 *
 * CUDA version
 */
__device__ void CUnewBI(void* dst, char* s, int sig) {
  int i = CUstrlen(s) - 1;
  int f = i;
  int j = 0;
  int c;
  int ssig = sig;

  //sign adjustment
  ((BigInteger*)dst)->k = 'i';

  //clean the array
  CUclean(dst);

  //iterate over the string and save data as integers
  for (; i >= 0; i--) {
    c = (int)(s[i] - 48);

    if (c >= 0 && c <= 9)
      ((BigInteger*)dst)->n[j++] = c;
    else
      if (s[i] == '-')
        ssig = -1;
  }

  //if a negative is sent, remove a position
  if (s[0] == '-')
    --f;

  ((BigInteger*)dst)->count = f;

  if (ssig == -1)
    ((BigInteger*)dst)->n[((BigInteger*)dst)->count] *= -1;
}

/*
 * CUclean. Cleans a BigInteger, moving 0 as value
 *
 * CUDA version
 */
__device__ static void CUclean(void* va) {
  int i = 0;

  for (i = 0; i < 1024; i++)
    ((BigInteger*)va)->n[i] = 0;

  ((BigInteger*)va)->count = 0;
}

/*
 * CUsMul.
 *
 * Simulates a *= b
 *
 * CUDA version
 */
__device__ void CUsMul(void* va, void* vb, void* vc, void* vd) {
  int sig;
  int i;
  int x;

  //clean aux vars (vc, vd)
  CUclean(vc);
  CUclean(vd);

  //get signum
  sig = CUsignum(((BigInteger*)va)->n[((BigInteger*)va)->count],
    ((BigInteger*)vb)->n[((BigInteger*)vb)->count]);

  //normalize
  if (sig == 1)
    ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
  else if (sig == 10)
    ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;
  else if (sig == 11) {
    ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;
    ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
  }

  //for each digit on vb
  for (i = 0; i <= ((BigInteger*)vb)->count; i++) {
    //perform partial product
    for (x = 0; x <= ((BigInteger*)va)->count; x++)
      ((BigInteger*)vc)->n[x] = ((BigInteger*)va)->n[x] * ((BigInteger*)vb)->n[i];

    //adjust length and carry
    ((BigInteger*)vc)->count = x - 1;
    CUcarryAdd(vc, 0, 0);

    //get offset
    CUpMul(i, vc);

    //add
    CUaddition(vd, vc);
  }

  //move result
  memcpy(va, vd, sizeof(BigInteger));

  //if signs are even, switch sign
  if (sig == 1 || sig == 10)
    ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;

  //denormalize
  if (sig == 1)
    ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
  else if (sig == 11)
    ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;

  //final recount
  CUrecount(va);
}

/*
 * CUpAdd
 *
 * Performs addition operation, having in count the number signs.
 * If signs are not the same, it performs subtract.
 *
 * CUDA version
 */
__device__ void CUpAdd(void* va, void* vb) {
  //get signum
  int sig = CUsignum(((BigInteger*)va)->n[((BigInteger*)va)->count],
    ((BigInteger*)vb)->n[((BigInteger*)vb)->count]);

  //normalize
  if (sig == 10)
    ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;
  else if (sig == 1)
    ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
  else if (sig == 11) {
    ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;
    ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
  }

  //if both signs are the same, add, else, subtract
  if (sig == 0 || sig == 11)
    CUaddition(va, vb);
  else
    CUsubtract(va, vb);

  if (sig == 10 || sig == 11)
    //switch back the sign
    ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;

  //denormalize
  if (sig == 1)
    ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
  else if (sig == 11)
    ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
}

/*
 * CUpSub.
 *
 * Performs subtraction operation, having in count the number signs.
 * If signs are not the same, it performs addition.
 *
 * CUDA version
 */
__device__ void CUpSub(void* va, void* vb) {
  int sig;

  //get signum
  sig = CUsignum(((BigInteger*)va)->n[((BigInteger*)va)->count],
    ((BigInteger*)vb)->n[((BigInteger*)vb)->count]);

  //normalize
  if (sig == 1)
    ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
  else if (sig == 10)
    ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;
  else if (sig == 11) {
    ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;
    ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
  }

  //if both signs are the same, subtract, else, add
  if (sig == 0 || sig == 11)
    CUsubtract(va, vb);
  else
    CUaddition(va, vb);

  if (sig == 10 || sig == 11)
    //switch back the sign
    ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;

  //denormalize
  if (sig == 1)
    ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
  else if (sig == 11)
    ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;

}

/*
 * CUsubrtact.
 *
 * Performs a -= b
 *
 * CUDA version
 */
__device__ static void CUsubtract(void* va, void* vb) {
  int i = 0;

  //subtract common part
  for (; i <= ((BigInteger*)vb)->count; i++)
    ((BigInteger*)va)->n[i] -= ((BigInteger*)vb)->n[i];

  //if last digit is negartive
  if (((BigInteger*)va)->n[((BigInteger*)va)->count] < 0)
    CUcarrySub(va, 1);
  else
    CUcarrySub(va, 0);
}

/*
 * CUcarrySub.
 *
 * Manages subtraction carry. If carryType = 0, the carry
 * manages as a += 10; else, it invert the sign (except last digit)
 *
 * CUDA version
 */
__device__ static void CUcarrySub(void* va, int carryType) {
  int i = 0;
  int acc = 0;

  if (carryType == 0) {
    for (; i <= ((BigInteger*)va)->count; i++) {
      //subtract carry to number
      ((BigInteger*)va)->n[i] -= acc;

      if (((BigInteger*)va)->n[i] < 0) {
        //normalize
        ((BigInteger*)va)->n[i] += 10;
        acc = 1;
      }
      else
        acc = 0;
    }
  } else {
    for (i = 0; i < ((BigInteger*)va)->count; i++)
      if (((BigInteger*)va)->n[i] < 0)
        //normalize
        ((BigInteger*)va)->n[i] = ((BigInteger*)va)->n[i] * -1;
  }

  //recount digits
  CUrecount(va);
}

/*
 * CUrecount.
 *
 * Count the digits, to check if count has to be diminished.
 *
 * CUDA version
 */
__device__ static void CUrecount(void* va) {
  while (((BigInteger*)va)->n[((BigInteger*)va)->count--] == 0);

  ++((BigInteger*)va)->count;

  if (((BigInteger*)va)->count < 0)
    ((BigInteger*)va)->count = 0;
}

/*
 * CUaddition.
 *
 * performs a += b
 *
 * CUDA version
 */
__device__ static void CUaddition(void* va, void* vb) {
  int limit;
  int min;
  int swap;
  int move;
  int i;

  //assume "a" has the bigger length
  limit = ((BigInteger*)va)->count;

  //assume "b" has the shortest length
  min = ((BigInteger*)vb)->count;

  //carry indicator
  move = 0;
  i = 0;

  //if assumption is wrong, rectify
  if (((BigInteger*)vb)->count > limit) {
    //swap limit and min
    swap = limit;
    limit = min;
    min = swap;

    move = 1;
  }

  //add all common digits
  for (; i <= min; i++)
    ((BigInteger*)va)->n[i] += ((BigInteger*)vb)->n[i];

  //move the even digits
  if (move == 1) {
    for (; i <= limit; i++)
      ((BigInteger*)va)->n[i] = ((BigInteger*)vb)->n[i];

    ((BigInteger*)va)->count = limit;
  }

  //carry management
  CUcarryAdd(va, 1, min);
}

/*
 * CUpMul.
 *
 * Ponderates vpart a pos number of positions, leaving pos 0's on the right side
 * That's a fancy way to say it performs vpart = vpart * 10^pos
 *
 * CUDA version
 */
__device__ static void CUpMul(int pos, void* vpart) {
  int i;

  i = ((BigInteger*)vpart)->count + pos;

  //generate offset
  for (; i >= pos; i--)
    ((BigInteger*)vpart)->n[i] = ((BigInteger*)vpart)->n[i - pos];

  //normalize offset positions
  for (i = 0; i < pos; i++)
    ((BigInteger*)vpart)->n[i] = 0;

  ((BigInteger*)vpart)->count += pos;
}

/*
 * CUcarryAdd.
 *
 * Manages addition carry.
 *
 * CUDA version
 */
__device__ void CUcarryAdd(void* va, int move, int min) {
  int i = 0;
  int acc;
  int limit;

  acc = 0;

  //move == 1 --> we know there's a non-common part. Min will be the common part threshold
  if (move == 1)
    limit = min;
  else
    limit = ((BigInteger*)va)->count;

  //iterate and manage carry on the common part
  for (; i <= limit; i++) {
    //add carry
    ((BigInteger*)va)->n[i] += acc;

    //as acc is int, we can divide by 10 and get the carry
    acc = ((BigInteger*)va)->n[i] / 10;

    if (acc > 0)
      //normalize
      ((BigInteger*)va)->n[i] = ((BigInteger*)va)->n[i] % 10;
  }

  if (move == 1) {
    //there's non-common part. Carry unitl acc = 0, as the non-common part is always normalized.
    while (acc > 0 && i <= ((BigInteger*)va)->count) {
      //add carry
      ((BigInteger*)va)->n[i] += acc;

      //as acc is int, we can divide by 10 and get the carry
      acc = ((BigInteger*)va)->n[i] / 10;

      if (acc > 0)
        //normalize
        ((BigInteger*)va)->n[i] = ((BigInteger*)va)->n[i] % 10;

      i++;
    }
  }

  //if there's a carry left, we move it to the end
  if (acc > 0)
    ((BigInteger*)va)->n[++(((BigInteger*)va)->count)] = acc;
}

/*
 * CUhardEquals.
 *
 * Compares two numbers.
 *   Return 0 if a = b
 *   Return 1 if a > b
 *   Return 2 if a < b
 *
 * CUDA version
 */
__device__ void CUhardEquals(void* va, void* vb, int* ret) {
  int i;
  int sig;

  //if pointer value is the same, they share value
  if (va == vb) {
    *ret = 0;
    return;
  }

  //get signum
  sig = CUsignum(((BigInteger*)va)->n[((BigInteger*)va)->count],
    ((BigInteger*)vb)->n[((BigInteger*)vb)->count]);

  if (sig == 1)
    //a >= 0, b < 0, so a > b
    *ret = 1;
  else if (sig == 10)
    //a < 0, b >= 0, so a < b
    *ret = 2;
  else {
    //they share signum. Manual comparation
    *ret = 0;

    if (((BigInteger*)va)->count < ((BigInteger*)vb)->count)
      //count(a) < count(b)
      *ret = 2;
    else if (((BigInteger*)va)->count > ((BigInteger*)vb)->count)
      //count(a) > count(b)
      *ret = 1;
    else {
      //count(a) = count(b). Manual compare
      for (i = ((BigInteger*)va)->count; i >= 0; i--) {
        if (((BigInteger*)va)->n[i] < ((BigInteger*)vb)->n[i])
          *ret = 2;
        else if ((((BigInteger*)va)->n[i] > ((BigInteger*)vb)->n[i]))
          *ret = 1;

        if (*ret > 0)
          break;
      }
    }

    if (sig == 11) {
      //both have negative sign. Switch return
      if (*ret == 2)
        *ret = 1;
      else if (*ret == 1)
        *ret = 2;
    }
  }
}

/*
 * CUsignum.
 *
 * Returns the quantity of negative data that are on an operation
 *   0 : None
 *   1 : b is negative
 *   10: a is negative
 *   11: a and b are negative
 *
 * CUDA version
 */
__device__ static int CUsignum(int a, int b) {
  int ret = 0;

  if (a < 0)
    ret = 10;

  if (b < 0)
    ++ret;

  return ret;
}

/*
 * CUsDvs.
 *
 * Performs operation a /= b
 *
 * CUDA version
 */
__device__ void CUsDvs(void* va, void* vb, void* xa, void* xb) {
  int sig;
  int comp;

  //initalize temp vars (xa, xb)
  CUnewBI(xa, "0", 0);
  CUnewBI(xb, "1", 0);


  //signum
  sig = CUsignum(((BigInteger*)va)->n[((BigInteger*)va)->count],
    ((BigInteger*)vb)->n[((BigInteger*)vb)->count]);

  //normalize
  if (sig == 1)
    ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
  else if (sig == 10)
    ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;
  else if (sig == 11) {
    ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;
    ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
  }

  CUhardEquals(va, vb, &comp);

  if (comp == 0) {
    //if a = b, a / b = 1
    CUhardEquals(va, xa, &comp);

    if (comp == 0)
      //if a = 0, then b = 0 (as a = b), then a / b = 0
      memcpy(va, xa, sizeof(BigInteger));
    else
      //otherwise, as a = b, a / b = 1
      memcpy(va, xb, sizeof(BigInteger));
  } else if (comp == 2) {
    //if a < b, then a / b = 0 (as we're on integer)
    memcpy(va, xa, sizeof(BigInteger));
  } else if (comp == 1) {
    //if a > b, then a / b = n
    CUhardEquals(vb, xb, &comp);

    if (comp != 0)
      //only search n if b != 1
      CUdivide(va, vb, xa, xb);
  }

  //if sign are even, we switch the sign
  if (sig == 1 || sig == 10)
    ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;

  //denormalize
  if (sig == 1)
    ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
  else if (sig == 11)
    ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
}

/*
 * CUdivide.
 *
 * Perform division via Bolzano
 *
 * CUDA version
 */
__device__ static void CUdivide(void* va, void* vb, void* xa, void* xb) {
  int len;
  int i = 0;
  int x = 0;
  int res = 0;
  int eq;

  len = ((BigInteger*)va)->count - ((BigInteger*)vb)->count;

  //initalize temp data (xa, xb)
  CUclean(xa);
  CUclean(xb);

  /*
   * Keep "b.count" first digits. If "b" has a single digit we don't move
   * anything, because we later will move a digit.
   */
  if (((BigInteger*)vb)->count > 0)
    for (; i < ((BigInteger*)vb)->count; i++)
      ((BigInteger*)xb)->n[((BigInteger*)vb)->count - i - 1] = ((BigInteger*)va)->n[((BigInteger*)va)->count - i];
  else
    ((BigInteger*)xb)->n[0] = 0;

  ((BigInteger*)xb)->count = ((BigInteger*)vb)->count - 1;

  if (((BigInteger*)xb)->count == -1)
    ++((BigInteger*)xb)->count;

  //for each digit we generated
  for (i = 0; i <= len; i++) {
    //make a temporal BI
    CUpAppend(xb, ((BigInteger*)va)->n[len - i]);

    /*
     * if b.len == 0, there's a bug on which the result of append has {n0} form with len = 1
     * when it should have {n} form with len = 0 (it only happens during first iteration)
     */
    if (((BigInteger*)xb)->count == 1 && ((BigInteger*)xb)->n[1] == 0)
      --((BigInteger*)xb)->count;

    for (x = 0; x < 10; x++) {
      if (((BigInteger*)xb)->n[((BigInteger*)xb)->count] == 0 && ((BigInteger*)xb)->count == 0) {
        //the result is 0
        res = x;
        x = 99;
      } else if (((BigInteger*)xb)->n[((BigInteger*)xb)->count] < 0) {
        //value is negative, we got the threshold
        CUaddition(xb, vb);
        res = (x - 1);
        x = 99;
      } else {
        //any other scenario. Check it
        CUhardEquals(xb, vb, &eq);

        if (eq == 2) {
          //xb < vb --> found
          res = x;
          x = 99;
        } else {
          //xb >= vb
          CUsubtract(xb, vb);
        }
      }
    }

    //move value
    ((BigInteger*)xa)->n[len - i] = res;
  }

  //move length
  ((BigInteger*)xa)->count = len;

  //copy result
  memcpy(va, (BigInteger*)xa, sizeof(BigInteger));

  //recount
  CUrecount(va);

  //avoid fake overflow error
  if (((BigInteger*)va)->n[((BigInteger*)va)->count + 1] > 0)
    ++((BigInteger*)va)->count;
}

/*
 * CUpAppend.
 *
 * Adds a digit at the end of the BigInteger
 *
 * CUDA version
 */
__device__ void CUpAppend(void* va, int b) {
  int i;

  for (i = ((BigInteger*)va)->count; i >= 0; i--)
    ((BigInteger*)va)->n[i + 1] = ((BigInteger*)va)->n[i];

  ((BigInteger*)va)->n[0] = b;
  ++((BigInteger*)va)->count;
}

/*
 * CUsub. Use it to subtract two numbers.
 *
 * CUDA version
 */
__device__ void CUsub(void* va, void* vb, void* vc) {
  int comp;
  int sig;

  CUhardEquals(va, vb, &comp);
  sig = CUsignum(((BigInteger*)va)->n[((BigInteger*)va)->count],
    ((BigInteger*)vb)->n[((BigInteger*)vb)->count]);

  //if |va| < |vb|
  if ((comp == 2 && sig < 11) || (comp == 1 && sig == 11)) {
    //vc = vb
    memcpy(vc, vb, sizeof(BigInteger));

    //vc -= va
    CUpSub(vc, va);

    //switch sign
    ((BigInteger*)vc)->n[((BigInteger*)vc)->count] *= -1;

    //move result
    memcpy(va, vc, sizeof(BigInteger));
  } else {
    //delegate on standard function
    CUpSub(va, vb);
  }
}

/*
 * CUsqrt. Use it to get the square root of a number.
 *
 * CUDA version
 */
__device__ void CUsqrt(void* va, void* vb, void* vc, void* vd) {
  char even = ((BigInteger*)va)->count % 2;
  int digits = ((BigInteger*)va)->count + 1;
  char fstep;
  int x = 0;
  int y = 0;
  int d = 0;
  int eq;
  char z;

  CUclean(vb);
  CUclean(vc);
  CUclean(vd);

  //create the first remainder
  if (even == 1) {
    //if even = 1 means that count is even, so there's a odd number of digits
    fstep = ((BigInteger*)va)->n[((BigInteger*)va)->count] * 10 + ((BigInteger*)va)->n[((BigInteger*)va)->count - 1];
    d = ((BigInteger*)va)->count - 2;
  } else {
    //else, there's an even number of digits
    fstep = ((BigInteger*)va)->n[((BigInteger*)va)->count];
    d = ((BigInteger*)va)->count - 1;
  }

  //get the root of fstep
  while (x * x++ <= fstep);
  if (--x * x > fstep)
    --x;

  //"x" will be the first digit of the root
  ((BigInteger*)vb)->n[((BigInteger*)vb)->count] = x;

  //save remainder on vc. It will always happens that fstep >= x
  ((BigInteger*)vc)->n[((BigInteger*)vc)->count] = fstep - (x * x);

  //recount
  CUrecount(vc);

  //use "digits" as counter
  if (digits == 1 || digits == 2) {
    //special case, for 1 and 2 digits, the result will have 1 digit only
    digits = 0;
  } else {
    digits = (digits / 2 + digits % 2) - 1;
  }

  //using "x" as temp counter, adding 1 as we already have the first digit
  x = 1;

  //while there's digits to move
  while (x <= digits) {
    //get the next two digits
    //first, we make an offset on vc
    y = ((BigInteger*)vc)->count + 2;

    for (; y >= 2; y--)
      ((BigInteger*)vc)->n[y] = ((BigInteger*)vc)->n[y - 2];

    //then we move correlative positions from va to vc
    ((BigInteger*)vc)->n[1] = ((BigInteger*)va)->n[d--];
    ((BigInteger*)vc)->n[0] = ((BigInteger*)va)->n[d--];

    //set vc length up to two
    ((BigInteger*)vc)->count += 2;

    //manage carry
    CUcarryAdd(vc, 0, 0);

    //copy current result (vb) to vd
    memcpy(vd, vb, sizeof(BigInteger));

    //calculate vb += vb. Doesn't matter the order here
    for (y = 0; y <= ((BigInteger*)vd)->count; y++)
      ((BigInteger*)vd)->n[y] += ((BigInteger*)vd)->n[y];

    //carry
    CUcarryAdd(vd, 0, 0);

    //find a number n = [1, 9] that satisfies 10vd + n^2 = vc
    //first, we create the value for n = 1
    y = ((BigInteger*)vd)->count + 1;

    for (; y >= 1; y--)
      ((BigInteger*)vd)->n[y] = ((BigInteger*)vd)->n[y - 1];

    ((BigInteger*)vd)->n[0] = 1;
    ++((BigInteger*)vd)->count;

    //init z, that will hold values for n
    z = 1;

    //assume we don't get to vb
    eq = 2;

    //searching z
    while (eq == 2) {
      //compare vd vs vb
      CUhardEquals(vd, vc, &eq);

      if (eq == 1) {
        //vd > vb. We need the previous digit
        //restore previous state
        if (z == 1) {
          //if z = 1, we're on the first iteration, so previous value will be 0
          CUnewBI(vd, "0", 0);
          --z;
        } else {
          //get z back
          --z;
        }
      } else if (eq == 2) {
        //vd < vb. The value we're searching is still not this
        //increase vd and z. Increase is 10vd + 2n + 1
        //get first part for vd

        //(vd * 20)
        for (y = 0; y <= ((BigInteger*)vb)->count; y++)
          ((BigInteger*)vd)->n[y + 1] += (((BigInteger*)vb)->n[y] * 2);

        //carry
        CUcarryAdd(vd, 0, 0);

        //+2n + 1
        ((BigInteger*)vd)->n[0] += (2 * z) + 1;

        //carry again
        CUcarryAdd(vd, 0, 0);

        ++z;
      }

      //if we iterated 10 times, value will be 9
      if (z == 10) {
        //back to 9
        --z;

        //force quit
        eq = 1;
      }
    }

    //restore previous value
    if (eq == 1) {
      //if we went too far, we get back to previous value
      //copy current result (vb) to vd
      memcpy(vd, vb, sizeof(BigInteger));

      //get vb += vb
      for (y = 0; y <= ((BigInteger*)vd)->count; y++)
        ((BigInteger*)vd)->n[y] += ((BigInteger*)vd)->n[y];

      //carry
      CUcarryAdd(vd, 0, 0);

      //create the rest of the value
      y = ((BigInteger*)vd)->count + 1;

      //offset
      for (; y >= 1; y--)
        ((BigInteger*)vd)->n[y] = ((BigInteger*)vd)->n[y - 1];

      //move z value
      ((BigInteger*)vd)->n[0] = z;
      ++((BigInteger*)vd)->count;

      //multiply by z
      for (y = 0; y <= ((BigInteger*)vd)->count; y++)
        ((BigInteger*)vd)->n[y] *= z;

      //carry
      CUcarryAdd(vd, 0, 0);
    }

    //move the digit
    /*
     * After the loop, we will have the digit to move on "z"; the partial result on vd; and the current remainder
     * on vc. We subtract vc -= vd (always will be vc >= vd) to update the remainder.
     */
    CUsubtract(vc, vd);

    //offseting vb
    y = ((BigInteger*)vb)->count + 1;

    for (; y >= 1; y--)
      ((BigInteger*)vb)->n[y] = ((BigInteger*)vb)->n[y - 1];

    //move z
    ((BigInteger*)vb)->n[0] = z;
    ++((BigInteger*)vb)->count;

    //next pair of digits
    x++;
  }

  //move resukt
  memcpy(va, vb, sizeof(BigInteger));
}

/*
 * CUstrlen. Mock-up for strlen as CUDA does not implement the function
 *
 * CUDA version
 */
__device__ static int CUstrlen(char* s) {
  int ret = 0;

  while (*s != '\0') {
    ++s;
    ++ret;
  }

  return ret;
}
#endif

/*
 * newBI.
 *
 * Generates a new BI from the input string (dst).
 * The load is made on reverse order to allow a simple growing mechanism.
 * If "sig" is -1, the number will be negative.
 */
void newBI(void* dst, const char* s, int sig) {
  int i = (int)strlen(s) - 1;
  int f = i;
  int j = 0;
  int c;
  int ssig = sig;

  //sign adjustment
  ((BigInteger*)dst)->k = 'i';

  //clean the array
  clean(dst);

  if (i > MAX_LENGTH + 1) {
    showError(1);
    return;
  }

  //iterate over the string and save data as integers
  for (; i >= 0; i--) {
    c = (int)(s[i] - 48);

    if (c >= 0 && c <= 9)
      ((BigInteger*)dst)->n[j++] = c;
    else
      if (s[i] == '-')
        ssig = -1;
      else {
        showError(2);
        return;
      }
  }

  //if a negative is sent, remove a position
  if (s[0] == '-')
    --f;

  ((BigInteger*)dst)->count = f;

  if (ssig == -1)
    ((BigInteger*)dst)->n[((BigInteger*)dst)->count] *= -1;
}

/*
 * validateBI
 *
 * It validates that all BI data are cohesive
 */
void validateBI(void* a) {
  int i = 0;

  //type validation
  if (((BigInteger*)a)->k != 'i') {
    showError(99);
    return;
  }

  //length validation
  if (((BigInteger*)a)->count < 0 || ((BigInteger*)a)->count > MAX_LENGTH) {
    showError(99);
    return;
  }

  //validate the rest of the digits, that can be positive or negative
  for (; i < MAX_LENGTH; i++) {
    if (((BigInteger*)a)->n[i] < -9 || ((BigInteger*)a)->n[i] > 9) {
      showError(99);
      return;
    }
  }
}

/*
 * getPoint
 *
 * Returns the decimal point cut for the division
 */
int getPoint() {
  return BI_point;
}

/*
 * getReturnCode
 *
 * Returns status code
 */
int getReturnCode() {
  return BIReturnCode;
}

/*
 * setReturnCode
 *
 * Sets the status code
 */
void setReturnCode(int k) {
  BIReturnCode = k;
}

/*
 * _BI_initialize
 *
 * Give value to util data
 */
#if BI_STANDALONE == 1 
static
#endif
void _BI_initialize() {
  int i = 0;

  //we create it from scratch as "clean" copies from _ZERO
  for (; i < MAX_LENGTH; i++)
    _ZERO.n[i] = 0;

  _ZERO.count = 0;
  _ZERO.k = 'i';

  newBI(&_ONE, "1", 0);
  newBI(&_TWO, "2", 0);
  newBI(&_THREE, "3", 0);
  newBI(&_FOUR, "4", 0);
  newBI(&_FIVE, "5", 0);
  newBI(&_SIX, "6", 0);
  newBI(&_SEVEN, "7", 0);
  newBI(&_EIGHT, "8", 0);
  newBI(&_NINE, "9", 0);
  newBI(&_TEN, "10", 0);
  newBI(&_HUND, "100", 0);
  newBI(&_MIN, "1", -1);
}

/*
 * BImemcpy
 *
 * Copies on dst pointer the useful data
 */
void BImemcpy(void* dst, int value) {
  if (value == 0)
    memcpy(dst, &_ZERO, sizeof(BigInteger));
  else if (value == 1)
    memcpy(dst, &_ONE, sizeof(BigInteger));
  else if (value == 2)
    memcpy(dst, &_TWO, sizeof(BigInteger));
  else if (value == 3)
    memcpy(dst, &_THREE, sizeof(BigInteger));
  else if (value == 4)
    memcpy(dst, &_FOUR, sizeof(BigInteger));
  else if (value == 5)
    memcpy(dst, &_FIVE, sizeof(BigInteger));
  else if (value == 6)
    memcpy(dst, &_SIX, sizeof(BigInteger));
  else if (value == 7)
    memcpy(dst, &_SEVEN, sizeof(BigInteger));
  else if (value == 8)
    memcpy(dst, &_EIGHT, sizeof(BigInteger));
  else if (value == 9)
    memcpy(dst, &_NINE, sizeof(BigInteger));
  else if (value == 10)
    memcpy(dst, &_TEN, sizeof(BigInteger));
  else if (value == 100)
    memcpy(dst, &_HUND, sizeof(BigInteger));
  else if (value == -1)
    memcpy(dst, &_MIN, sizeof(BigInteger));
  else
    memcpy(dst, &_ZERO, sizeof(BigInteger));
}

/*
 * pAppend.
 *
 * Adds a digit at the end of the BigInteger
 */
static void pAppend(void* va, int b) {
  int i;

  for (i = ((BigInteger*)va)->count; i >= 0; i--)
    ((BigInteger*)va)->n[i + 1] = ((BigInteger*)va)->n[i];

  ((BigInteger*)va)->n[0] = b;
  ++((BigInteger*)va)->count;
}

/*
 * signum.
 *
 * Returns the quantity of negative data that are on an operation
 *   0 : None
 *   1 : b is negative
 *   10: a is negative
 *   11: a and b are negative
 */
static int signum(int a, int b) {
  int ret = 0;

  if (a < 0)
    ret = 10;

  if (b < 0)
    ++ret;

  return ret;
}

/*
 * hardEquals.
 *
 * Compares two numbers.
 *   Return 0 if a = b
 *   Return 1 if a > b
 *   Return 2 if a < b
 */
#if BI_STANDALONE == 1 
static
#endif
void hardEquals(void* va, void* vb, int* ret) {
  int i;
  int sig;

  //if pointer value is the same, they share value
  if (va == vb) {
    *ret = 0;
    return;
  }

  //get signum
  sig = signum(((BigInteger*)va)->n[((BigInteger*)va)->count],
    ((BigInteger*)vb)->n[((BigInteger*)vb)->count]);

  if (sig == 1)
    //a >= 0, b < 0, so a > b
    *ret = 1;
  else if (sig == 10)
    //a < 0, b >= 0, so a < b
    *ret = 2;
  else {
    //they share signum. Manual comparation
    *ret = 0;

    if (((BigInteger*)va)->count < ((BigInteger*)vb)->count)
      //count(a) < count(b)
      *ret = 2;
    else if (((BigInteger*)va)->count > ((BigInteger*)vb)->count)
      //count(a) > count(b)
      *ret = 1;
    else {
      //count(a) = count(b). Manual compare
      for (i = ((BigInteger*)va)->count; i >= 0; i--) {
        if (((BigInteger*)va)->n[i] < ((BigInteger*)vb)->n[i])
          *ret = 2;
        else if ((((BigInteger*)va)->n[i] > ((BigInteger*)vb)->n[i]))
          *ret = 1;

        if (*ret > 0)
          break;
      }
    }

    if (sig == 11) {
      //both have negative sign. Switch return
      if (*ret == 2)
        *ret = 1;
      else if (*ret == 1)
        *ret = 2;
    }
  }
}

/*
 * pAdd
 *
 * Performs addition operation, having in count the number signs.
 * If signs are not the same, it performs subtract.
 */
#if BI_STANDALONE == 1
static
#endif
 void pAdd(void* va, void* vb, void* m) {
  if (((memory*)m)->vt == NULL) {
    showError(11);
    return;
  }

  if (va == vb) {
    //add(a, a); delegate to mul(a, 2)
    sMul(va, ((memory*)m)->vt, m);
  } else {
    //add(a, b)
    int sig = signum(((BigInteger*)va)->n[((BigInteger*)va)->count],
      ((BigInteger*)vb)->n[((BigInteger*)vb)->count]);

    //normalize operators
    if (sig == 10)
      //a negative, b positive. Change "a" sign and perform addition
      ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;
    else if (sig == 1)
      //b negative, a positive. Change "b" sign and perform addition
      ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
    else if (sig == 11) {
      //a negative, b negative. Change signs and perform addition
      ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;
      ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
    }

    //if both signs are the same we add, else we subtract
    if (sig == 0 || sig == 11)
      addition(va, vb);
    else
      //can't call subtraction as pSub does essential validations towards subtraction (i.e length of a vs b). #stackloop
      pSub(va, vb, m);

    if (sig == 10 || sig == 11)
      //switch back the sign
      ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;

    //denormalize b
    if (sig == 1)
      ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
    else if (sig == 11)
      ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
  }
}

/*
 * addition.
 *
 * performs a += b
 */
static void addition(void* va, void* vb) {
  int limit;
  int min;
  int swap;
  int move;
  int i;

  //assume "a" has the bigger lenght 
  limit = ((BigInteger*)va)->count;

  //assume "b" has the shortest length
  min = ((BigInteger*)vb)->count;

  //carry indicator 
  move = 0;
  i = 0;

  //if the assumption is wrong, rectify
  if (((BigInteger*)vb)->count > limit) {
    //switch limit and min
    swap = limit;
    limit = min;
    min = swap;

    move = 1;
  }

  //add all the common digits
  for (; i <= min; i++)
    ((BigInteger*)va)->n[i] += ((BigInteger*)vb)->n[i];

  //move the even digits
  if (move == 1) {
    for (; i <= limit; i++)
      ((BigInteger*)va)->n[i] = ((BigInteger*)vb)->n[i];

    ((BigInteger*)va)->count = limit;
  }

  //carry management
  carryAdd(va, 1, min);
}

/*
 * carryAdd.
 *
 * Manages addition carry.
 */
#if BI_STANDALONE == 1
static
#endif
 void carryAdd(void* va, int move, int min) {
  int i = 0;
  int acc;
  int limit;

  acc = 0;

  //move == 1 --> we know there's a non-common part. Min will be the common part threshold
  if (move == 1)
    limit = min;
  else
    limit = ((BigInteger*)va)->count;

  //iterate and manage carry on the common part
  for (; i <= limit; i++) {
    //add carry
    ((BigInteger*)va)->n[i] += acc;

    //as acc is int, we can divide by 10 and get the carry
    acc = ((BigInteger*)va)->n[i] / 10;

    if (acc > 0)
      //normalize the number
      ((BigInteger*)va)->n[i] = ((BigInteger*)va)->n[i] % 10;
  }

  if (move == 1) {
    //there's non-common part. Carry unitl acc = 0, as the non-common part is always normalized.
    while (acc > 0 && i <= ((BigInteger*)va)->count) {
      //add carry
      ((BigInteger*)va)->n[i] += acc;

      //as acc is int, we can divide by 10 and get the carry
      acc = ((BigInteger*)va)->n[i] / 10;

      if (acc > 0)
        //normalize the number
        ((BigInteger*)va)->n[i] = ((BigInteger*)va)->n[i] % 10;

      i++;
    }
  }

  //if there's a carry left, we move it to the end
  if (acc > 0) {
    if (((BigInteger*)va)->count == MAX_LENGTH) {
      showError(1);
      return;
    } else
      ((BigInteger*)va)->n[++(((BigInteger*)va)->count)] = acc;
  }
}

/*
 * pSub.
 *
 * Performs subtraction operation, having in count the number signs.
 * If signs are not the same, it performs addition.
 */
#if BI_STANDALONE == 1
static
#endif
 void pSub(void* va, void* vb, void* m) {
  int sig;
  int comp;
  int sp = 0;

  if (((memory*)m)->stmp == NULL) {
    showError(10);
    return;
  }

  if (va == vb) {
    //sub(a, a); result = 0
    BImemcpy(va, 0);
  } else {
    //sub(a, b);
    hardEquals(va, vb, &comp);

    sig = signum(((BigInteger*)va)->n[((BigInteger*)va)->count],
      ((BigInteger*)vb)->n[((BigInteger*)vb)->count]);

    //if both are negative and comp = 1 means that a < b
    if ((comp == 2 && sig < 11) || (comp == 1 && sig == 11)) {
      sp = 1;

      memcpy(((memory*)m)->stmp, vb, sizeof(BigInteger)); //stmp = b

      //switch va and vb
      memcpy(vb, va, sizeof(BigInteger)); //vb = va
      memcpy(va, ((memory*)m)->stmp, sizeof(BigInteger)); //va = tmp

      //recalculate signum
      sig = signum(((BigInteger*)va)->n[((BigInteger*)va)->count],
        ((BigInteger*)vb)->n[((BigInteger*)vb)->count]);
    } else if (comp == 0)
      BImemcpy(va, 0);
    
    //normalize sign
    if (sig == 1)
      ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
    else if (sig == 10)
      ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;
    else if (sig == 11) {
      ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;
      ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
    }

    //if they have same sign we subtract; else we add
    if (sig == 0 || sig == 11)
      subtract(va, vb);
    else
      addition(va, vb);

    if (sig == 10 || sig == 11)
      //on this cases, we switch sign
      ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;

    //denormalize b
    if (sig == 1)
      ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
    else if (sig == 11)
      ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;

    if (sp == 1) {
      //special things to be done
      //switch sign
      ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;

      //retrieve vb original value
      memcpy(vb, ((memory*)m)->stmp, sizeof(BigInteger));
    }
  }
}

/*
 * subrtact.
 *
 * Performs a -= b
 */
static void subtract(void* va, void* vb) {
  int i = 0;

  //subtract common digits
  for (; i <= ((BigInteger*)vb)->count; i++)
    ((BigInteger*)va)->n[i] -= ((BigInteger*)vb)->n[i];

  //if last digit is negative
  if (((BigInteger*)va)->n[((BigInteger*)va)->count] < 0)
    carrySub(va, 1);
  else
    carrySub(va, 0);
}

/*
 * carrySub.
 *
 * Manages subtraction carry. If carryType = 0, the carry
 * manages as a += 10; else, it invert the sign (except last digit)
 */
static void carrySub(void* va, int carryType) {
  int i = 0;
  int acc = 0;

  if (carryType == 0) {
    for (; i <= ((BigInteger*)va)->count; i++) {
      //subtract carry to number
      ((BigInteger*)va)->n[i] -= acc;

      if (((BigInteger*)va)->n[i] < 0) {
        //normalize the number
        ((BigInteger*)va)->n[i] += 10;
        acc = 1;
      } else
        acc = 0;
    }
  } else {
    for (i = 0; i < ((BigInteger*)va)->count; i++)
      if (((BigInteger*)va)->n[i] < 0)
        //normalize the number
        ((BigInteger*)va)->n[i] = ((BigInteger*)va)->n[i] * -1;
  }

  //count the digits again
  recount(va);
}

/*
 * recount.
 *
 * Count the digits, to check if count has to be diminished.
 */
static void recount(void* va) {
  while (((BigInteger*)va)->n[((BigInteger*)va)->count--] == 0);

  ++((BigInteger*)va)->count;

  if (((BigInteger*)va)->count < 0)
    ((BigInteger*)va)->count = 0;
}

/*
 * sMul.
 *
 * Simulates a *= b
 */
#if BI_STANDALONE == 1
static
#endif
 void sMul(void* va, void* vb, void* m) {
  int sig;
  int i = 0;
  int x;
  int comp;
  int calc = 0;

  if (((memory*)m)->biBIT == NULL || ((memory*)m)->mzero == NULL || ((memory*)m)->mone == NULL ||
    ((memory*)m)->mpart == NULL || ((memory*)m)->mret == NULL || ((memory*)m)->mtmp == NULL) {
    showError(3);
    return;
  }

  if (va == vb) {
    //mul(a, a)
    //we copy it to tmp to work without collapsing the data
    memcpy(((memory*)m)->mtmp, va, sizeof(BigInteger));

    sMul(va, ((memory*)m)->mtmp, m);
  } else {
    //mul(a, b)
    BImemcpy(((memory*)m)->mzero, 0);
    BImemcpy(((memory*)m)->mret, 0);
    BImemcpy(((memory*)m)->mone, 1);

    //if a = 0, no need to calculate
    hardEquals(va, ((memory*)m)->mzero, &comp);

    if (comp == 0)
      calc = 1;

    //if a = 0, no need to calculate
    hardEquals(vb, ((memory*)m)->mzero, &comp);

    if (comp == 0) {
      memcpy(va, ((memory*)m)->mzero, sizeof(BigInteger));
      calc = 1;
    }

    //normalize numbers
    sig = signum(((BigInteger*)va)->n[((BigInteger*)va)->count],
      ((BigInteger*)vb)->n[((BigInteger*)vb)->count]);

    if (sig == 1)
      ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
    else if (sig == 10)
      ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;
    else if (sig == 11) {
      ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;
      ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
    }

    //if |a| = 1, then a * b = b (sign will be later normalized)
    hardEquals(va, ((memory*)m)->mone, &comp);

    if (comp == 0) {
      memcpy(va, vb, sizeof(BigInteger));
      calc = 1;
    }

    //if |b| = 1, then a * b = a (sign will be later normalized)
    hardEquals(vb, ((memory*)m)->mone, &comp);

    if (comp == 0)
      calc = 1;

    //initialize BIT
    memcpy(&((BIT*)((memory*)m)->biBIT)->BI[0], ((memory*)m)->mzero, sizeof(BigInteger));
    memcpy(&((BIT*)((memory*)m)->biBIT)->BI[1], va, sizeof(BigInteger));

    ((BIT*)((memory*)m)->biBIT)->status[0] = 1;
    ((BIT*)((memory*)m)->biBIT)->status[1] = 1;
    ((BIT*)((memory*)m)->biBIT)->status[2] = 0;
    ((BIT*)((memory*)m)->biBIT)->status[3] = 0;
    ((BIT*)((memory*)m)->biBIT)->status[4] = 0;
    ((BIT*)((memory*)m)->biBIT)->status[5] = 0;
    ((BIT*)((memory*)m)->biBIT)->status[6] = 0;
    ((BIT*)((memory*)m)->biBIT)->status[7] = 0;
    ((BIT*)((memory*)m)->biBIT)->status[8] = 0;
    ((BIT*)((memory*)m)->biBIT)->status[9] = 0;

    //if the number is not calculated (a = 0, b = 0, |a| = 1, |b| = 1), we calculate it
    if (calc == 0) {
      //perform partial product for each digit of b
      for (i = 0; i <= ((BigInteger*)vb)->count; i++) {
        //validate if BIT[n] exists
        if (((BIT*)((memory*)m)->biBIT)->status[((BigInteger*)vb)->n[i]] == 0) {
          clean(((memory*)m)->mpart);

          //we don't have the BIT loaded, so we calculate it
          for (x = 0; x <= ((BigInteger*)va)->count; x++)
            ((BigInteger*)((memory*)m)->mpart)->n[x] = ((BigInteger*)va)->n[x] * ((BigInteger*)vb)->n[i];

          ((BigInteger*)((memory*)m)->mpart)->count = x - 1;
          carryAdd(((memory*)m)->mpart, 0, 0);

          //move the value to corresponding BIT
          memcpy(&((BIT*)((memory*)m)->biBIT)->BI[((BigInteger*)vb)->n[i]], ((memory*)m)->mpart, sizeof(BigInteger));
          ((BIT*)((memory*)m)->biBIT)->status[((BigInteger*)vb)->n[i]] = 1;
        } else
          //we have a loaded BIT, so we copy it
          memcpy(((memory*)m)->mpart, &((BIT*)((memory*)m)->biBIT)->BI[((BigInteger*)vb)->n[i]], sizeof(BigInteger));

        //ponderate the result with "i" 0's
        pMul(i, ((memory*)m)->mpart);

        //add it
        addition(((memory*)m)->mret, ((memory*)m)->mpart);

        //init again
        BImemcpy(((memory*)m)->mpart, 0);
      }

      //move the result
      memcpy(va, ((memory*)m)->mret, sizeof(BigInteger));
    }

    //if signs are even, we switch it
    if (sig == 1 || sig == 10)
      ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;

    //denormalize vb
    if (sig == 1)
      ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
    else if (sig == 11)
      ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
  }
}

/*
 * pMul.
 *
 * Ponderates vpart a pos number of positions, leaving pos 0's on the right side
 * That's a fancy way to say it performs vpart = vpart * 10^pos
 */
static void pMul(int pos, void* vpart) {
  int i;

  i = ((BigInteger*)vpart)->count + pos;

  if (i >= MAX_LENGTH) {
    showError(1);
    return;
  }

  //generate offset
  for (; i >= pos; i--)
    ((BigInteger*)vpart)->n[i] = ((BigInteger*)vpart)->n[i - pos];

  //normalize offset positions
  for (i = 0; i < pos; i++)
    ((BigInteger*)vpart)->n[i] = 0;

  ((BigInteger*)vpart)->count += pos;
}

/*
 * sDvs.
 *
 * Performs operation a /= b
 */
#if BI_STANDALONE == 1
static
#endif
 void sDvs(void* va, void* vb, void* m) {
  int sig;
  int comp;

  if (((memory*)m)->dtmp == NULL || ((memory*)m)->done == NULL) {
    showError(4);
    return;
  }

  //init decimal point
  BI_point = -1;

  if (va == vb) {
    //dvs(a, a)
    BImemcpy(va, 1);
  } else {
    //dvs(a, b)

    BImemcpy(((memory*)m)->dtmp, 0);
    BImemcpy(((memory*)m)->done, 1);

    sig = signum(((BigInteger*)va)->n[((BigInteger*)va)->count],
      ((BigInteger*)vb)->n[((BigInteger*)vb)->count]);

    //normalize numbers
    if (sig == 1)
      ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
    else if (sig == 10)
      ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;
    else if (sig == 11) {
      ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;
      ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
    }

    hardEquals(va, vb, &comp);

    if (comp == 0) {
      //if a = b, a / b = 1
      hardEquals(va, ((memory*)m)->dtmp, &comp);

      //with this we indicate there's no decimal point
      BI_point = -2;

      if (comp == 0)
        //if a = 0, then b = 0 (as a = b), then a / b = 0
        memcpy(va, ((memory*)m)->dtmp, sizeof(BigInteger));
      else
        //otherwise, as a = b, a / b = 1
        memcpy(va, ((memory*)m)->done, sizeof(BigInteger));
    } else if (comp == 2)
      //if a < b, then a / b = 0 (as we're on integer)
      memcpy(va, ((memory*)m)->dtmp, sizeof(BigInteger));
    else if (comp == 1) {
      //if a > b, then a / b = n
      hardEquals(vb, ((memory*)m)->done, &comp);

      if (comp != 0)
        //only search n if b != 1
        divide(va, vb, m);
    }

    //if sign are even, we switch the sign
    if (sig == 1 || sig == 10)
      ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;

    //denormalize vb
    if (sig == 1)
      ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
    else if (sig == 11)
      ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
  }
}

/*
 * divide.
 *
 * Perform division via Bolzano
 */
static void divide(void* va, void* vb, void* m) {
  int len;
  int i = 0;
  int x = 0;
  int eq;
  int currentBIT;
  int res = 0;
  int added;
  int dlen;

  if (((memory*)m)->biBIT == NULL || ((memory*)m)->dTemp == NULL || ((memory*)m)->dret == NULL ||
    ((memory*)m)->biTemp == NULL) {
    showError(5);
    return;
  }

  BImemcpy(((memory*)m)->dret, 0);

  len = ((BigInteger*)va)->count - ((BigInteger*)vb)->count;

  //init BIT
  BImemcpy(&((BIT*)((memory*)m)->biBIT)->BI[0], 0);
  memcpy(&((BIT*)((memory*)m)->biBIT)->BI[1], vb, sizeof(BigInteger));

  ((BIT*)((memory*)m)->biBIT)->status[0] = 1;
  ((BIT*)((memory*)m)->biBIT)->status[1] = 1;
  ((BIT*)((memory*)m)->biBIT)->status[2] = 0;
  ((BIT*)((memory*)m)->biBIT)->status[3] = 0;
  ((BIT*)((memory*)m)->biBIT)->status[4] = 0;
  ((BIT*)((memory*)m)->biBIT)->status[5] = 0;
  ((BIT*)((memory*)m)->biBIT)->status[6] = 0;
  ((BIT*)((memory*)m)->biBIT)->status[7] = 0;
  ((BIT*)((memory*)m)->biBIT)->status[8] = 0;
  ((BIT*)((memory*)m)->biBIT)->status[9] = 0;

  currentBIT = 1;

  //init dTemp
  BImemcpy(((memory*)m)->dTemp, 0);

  /*
   * Keep "b.count" first digits. If "b" has a single digit we don't move
   * anything, because we later will move a digit.
   */
  if (((BigInteger*)vb)->count > 0)
    for (; i < ((BigInteger*)vb)->count; i++)
      ((BigInteger*)((memory*)m)->dTemp)->n[((BigInteger*)vb)->count - i - 1] = ((BigInteger*)va)->n[((BigInteger*)va)->count - i];
  else
    ((BigInteger*)((memory*)m)->dTemp)->n[0] = 0;

  ((BigInteger*)((memory*)m)->dTemp)->count = ((BigInteger*)vb)->count - 1;

  if (((BigInteger*)((memory*)m)->dTemp)->count == -1)
    ++((BigInteger*)((memory*)m)->dTemp)->count;

  //keep dlen for double calculation
  dlen = len;

  //init decimal point
  BI_point = 0;

  if (((BigInteger*)va)->k == 'd') {
    if (len == 0)
      BI_point = len;
    else
      BI_point = len - 1;

    len = MAX_LENGTH - 1;

    if (((BigInteger*)vb)->count == 0)
      len = MAX_LENGTH - 2;
    else
      len = MAX_LENGTH - 1;
  }

  //for each digit we generated
  for (i = 0; i <= len; i++) {
    //make a temporal BI
    if (i <= dlen)
      //there are still digits
      pAppend(((memory*)m)->dTemp, ((BigInteger*)va)->n[dlen - i]);
    else
      //no more digits, just carry a 0
      pAppend(((memory*)m)->dTemp, 0);

    /*
     * if b.len == 0, there's a bug on which the result of append has {n0} form with len = 1
     * when it should have {n} form with len = 0 (it only happens during first iteration)
     */
    if (((BigInteger*)((memory*)m)->dTemp)->count == 1 && ((BigInteger*)((memory*)m)->dTemp)->n[1] == 0)
      --((BigInteger*)((memory*)m)->dTemp)->count;

    hardEquals(((memory*)m)->dTemp, &((BIT*)((memory*)m)->biBIT)->BI[currentBIT], &eq);

    x = currentBIT;

    if (eq == 1) {
      //if dTemp > BIT[x], we start from that value and keep going

      //retrieve the last BI
      memcpy(((memory*)m)->biTemp, &((BIT*)((memory*)m)->biBIT)->BI[x], sizeof(BigInteger));
      added = 0;

      for (; x < 10; x++) {
        //add the base (vb)
        pAdd(((memory*)m)->biTemp, vb, m);

        //add it to BIT, if there's still space (we move it before validation)
        if (currentBIT < 9) {
          memcpy(&((BIT*)((memory*)m)->biBIT)->BI[++currentBIT], ((memory*)m)->biTemp, sizeof(BigInteger));
          ((BIT*)((memory*)m)->biBIT)->status[currentBIT] = 1;
          added = 1;
        } else
          added = 0;

        //check if it fits
        hardEquals(((memory*)m)->dTemp, ((memory*)m)->biTemp, &eq);

        if (eq == 0) {
          //if dTemp = temp
          res = currentBIT;
          x = 99;
        } else if (eq == 2) {
          /*
           * if dTemp < temp, as Bolzano theoreme, we already went thru dTemp = temp.
           * so, the correct value was the previous.
           *
           * If we didn't reach the end of the table (added == 1), the index will be the previous
           * If we reached the end of the table (added == 0), the index will be the current
           */
          if (added == 1)
            res = (currentBIT - 1);
          else
            res = currentBIT;

          x = 99;
        }
      }
    } else if (eq == 0)
      //if dTemp = BIT[x], we already have the value
      res = currentBIT;
    else if (eq == 2) {
      //if dTemp < BIT[x], start for that value and go thru 0
      --x;

      for (; x >= 0; x--) {
        //check if it fits
        hardEquals(((memory*)m)->dTemp, &((BIT*)((memory*)m)->biBIT)->BI[x], &eq);

        if (eq == 0) {
          //if dTemp = temp
          res = x;
          x = -99;
        } else if (eq == 1) {
          /*
           * if dtemp > temp, as Bolzano theoreme, we already went thru dTemp = temp,
           * so, correct value is the next one.
           */
          res = x;
          x = -99;
        }
      }
    }

    //subtract. Can't call subtraction as pSub does main validations. #stackloop
    pSub(((memory*)m)->dTemp, &((BIT*)((memory*)m)->biBIT)->BI[res], m);

    if (i <= dlen)
      ((BigInteger*)((memory*)m)->dret)->n[len - i] = res;
    else {
      //we're on the decimal zone. Check if it worth to add the data
      if (((BigInteger*)((memory*)m)->dTemp)->count == 0 && ((BigInteger*)((memory*)m)->dTemp)->n[0] == 0) {
        //we found a division with remainder = 0. There's no more data to treat and we finish
        i = len + 1;
      } else
        //There's still data to treat, add it
        ((BigInteger*)((memory*)m)->dret)->n[len - i] = res;
    }
  }

  ((BigInteger*)((memory*)m)->dret)->count = len;

  memcpy(va, ((memory*)m)->dret, sizeof(BigInteger));

  recount(va);

  //avoid fake overflow error
  if (((BigInteger*)va)->n[((BigInteger*)va)->count + 1] > 0)
    ++((BigInteger*)va)->count;
}

/*
 * sNqrt.
 *
 * Perform nth root of a.
 */
#if BI_STANDALONE == 1
static
#endif
 void sNqrt(void* va, int n, void* m) {
  int lmax = 0;
  int isEq = 0;

  if (((memory*)m)->sret == NULL || ((memory*)m)->sraw == NULL ||
    ((memory*)m)->sbase == NULL || ((memory*)m)->szero == NULL) {
    showError(9);
    return;
  }

  BImemcpy(((memory*)m)->sbase, 0);
  BImemcpy(((memory*)m)->szero, 0);

  //if root index is 0, return 0
  if (n <= 0) {
    memcpy(va, ((memory*)m)->szero, sizeof(BigInteger));

    return;
  }

  //as default behaviour, the nth root of a number will have half the digits #ToDo: review/optimize
  lmax = ((BigInteger*)va)->count / 2;

  //creat a base
  ((BigInteger*)((memory*)m)->sbase)->n[lmax] = 1;
  ((BigInteger*)((memory*)m)->sbase)->count = lmax;

  //use Bolzano to get an approximation
  memcpy(((memory*)m)->sret, ((memory*)m)->sbase, sizeof(BigInteger));
  memcpy(((memory*)m)->sraw, ((memory*)m)->sbase, sizeof(BigInteger));

  //calculate the power
  sBipow(((memory*)m)->sret, n, m);
  hardEquals(((memory*)m)->sret, va, &isEq);

  while (isEq != 0) {
    while (isEq == 2) {
      //ret < a. Increase and try again until overflow
      addition(((memory*)m)->sraw, ((memory*)m)->sbase);

      memcpy(((memory*)m)->sret, ((memory*)m)->sraw, sizeof(BigInteger));

      sBipow(((memory*)m)->sret, n, m);
      hardEquals(((memory*)m)->sret, va, &isEq);
    }

    //once here, it will always happens that ret >= a
    if (isEq == 1) {
      //if a > ret, get back 1 base position and go adjust the previous position. #stackloop
      pSub(((memory*)m)->sraw, ((memory*)m)->sbase, m);

      ((BigInteger*)((memory*)m)->sbase)->n[lmax] = 0;
      --lmax;

      if (lmax >= 0) {
        //if lmax is greater than 0, we still have digits to pivot
        ((BigInteger*)((memory*)m)->sbase)->n[lmax] = 1;
        --((BigInteger*)((memory*)m)->sbase)->count;

        //calculate power and compare (always will be a < ret, because a = ret case won't enter this if).
        hardEquals(((memory*)m)->sbase, ((memory*)m)->szero, &isEq);

        //if base > 0, we still have to iterate
        if (isEq == 1)
          isEq = 2;
        else
          //on the opposite case, we reached maximum possible aproximation
          isEq = 0;
      } else
        //if lmax is lower than 0, we reached maximum possible approximation
        isEq = 0;
    }
  }

  memcpy(va, ((memory*)m)->sraw, sizeof(BigInteger));
}

/*
 * sBipow.
 *
 * Performs a ^= p
 */
#if BI_STANDALONE == 1
static
#endif
void sBipow(void* va, int p, void* m) {
  int d2b[10];
  int d2bi = 0;
  int x = 0;
  int t = p;
  int i = 0;
  int sig = 0;

  if (((memory*)m)->bres == NULL || ((memory*)m)->btmp == NULL) {
    showError(6);
    return;
  }

  if (p < 0) {
    BImemcpy(va, 0);

    return;
  }

  BImemcpy(((memory*)m)->bres, 1);

  /*
   *  Factorize the power on base-2 so we can improve performance
   *  of the operation.
   *
   *  How it works
   *   - Convert exponent on base-2, and store data on an array
   *
   *   - For each iteration, we keep calculating a^2^i, that means
   *     a^0, a^2, a^4, a^8, a^16, ...
   *
   *   - If d2b position is 1, multiply, and this way we get the result
   *
   *     Example: 8^12
   *       12 = 1100 (but keeps stored as 0011) ==> 4 + 8
   *       8^12 = 8^4 * 8^8 = 8^(4+8)
   *
   *     With this way, we only do 8 multiplications instead of 12 (on this
   *     example). On larger numbers, the difference is still greater
   */
   //decimal to binary conversion
  while (t > 0 && d2bi < 10) {
    //divide by two
    x = t / 2;

    //keep the remainder on d2b
    d2b[d2bi++] = (t - x * 2);

    //reassign value
    t = x;
  }

  //if there's a really big number, we throw an error
  if (t > 0 && d2bi >= 10) {
    showError(8);
    return;
  }

  //normalize va
  if (((BigInteger*)va)->n[((BigInteger*)va)->count] < 0) {
    sig = 1;
    ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;
  }

  //iterate over binary number
  for (; i < d2bi; i++) {
    //calculate (a^(2^i))
    if (i == 0)
      memcpy(((memory*)m)->btmp, va, sizeof(BigInteger));
    else
      sMul(((memory*)m)->btmp, ((memory*)m)->btmp, m);

    if (d2b[i] == 1)
      sMul(((memory*)m)->bres, ((memory*)m)->btmp, m);
  }

  //revert va if exponent is even
  if (sig == 1 && p % 2 == 1)
    ((BigInteger*)va)->n[((BigInteger*)va)->count] *= -1;

  memcpy(va, ((memory*)m)->bres, sizeof(BigInteger));
}

#if BI_STANDALONE == 1
/*
 * add. Use it to add two numbers.
 */
void add(void* va, void* vb, void* m) {
  //validate data before treating
  if (validate == 1) {
    validateBI(va);
    validateBI(vb);
  }

  //delegate on static function
  pAdd(va, vb, m);
}

/*
 * sub. Use it to subtract two numbers.
 */
void sub(void* va, void* vb, void* m) {
  //validate data before treating
  if (validate == 1) {
    validateBI(va);
    validateBI(vb);
  }

  //delegate on static function
  pSub(va, vb, m);
}

/*
 * mul. Use it to multiply two numbers.
 */
void mul(void* va, void* vb, void* m) {
  //validate data before treating
  if (validate == 1) {
    validateBI(va);
    validateBI(vb);
  }

  //delegate on static function
  sMul(va, vb, m);
}

/*
 * dvs. Use it to divide two numbers.
 */
void dvs(void* va, void* vb, void* m) {
  //validate data before treating
  if (validate == 1) {
    validateBI(va);
    validateBI(vb);
  }

  //init the decimal point
  BI_point = 0;

  //delegate on static function
  sDvs(va, vb, m);
}

/*
 * nqrt. Use it to get the nth root of a number.
 */
void nqrt(void* va, int n, void* m) {
  //validate data before treating
  if (validate == 1)
    validateBI(va);

  //delegate on static function
  sNqrt(va, n, m);
}

/*
 * bipow. Use it to get the "p" power of a number.
 */
void bipow(void* va, int p, void* m) {
  //validate data before treating
  if (validate == 1)
    validateBI(va);

  if (p == 0)
    //n^0 = 1
    BImemcpy(va, 1);
  else if (p == 1)
    //n^1 = n
    return;
  else
    sBipow(va, p, m);
}

/*
 * mod. Use it to get the "b" module of a number.
 */
void mod(void* va, void* vb, void* m) {
  //validate data before treating
  if (validate == 1) {
    validateBI(va);
    validateBI(vb);
  }

  //delegate on static function
  sDvs(va, vb, m);

  //copy the remainder of the divison
  memcpy(va, ((memory*)m)->dTemp, sizeof(BigInteger));
}

/*
 * toString. Gets the string representation of a BigInteger
 */
void toString(void* vb, char* dst) {
  int i = 0;
  int m = ((BigInteger*)vb)->count;
  int sig = 0;

  //validate data before treating
  if (validate == 1)
    validateBI(vb);

  //if first digit is negative, we turn it positive and flag the character
  if (((BigInteger*)vb)->n[m] < 0) {
    ((BigInteger*)vb)->n[m] *= -1;
    dst[i++] = '-';
    sig = 1;
  }

  for (; m >= 0; m--)
    dst[i++] = (char)(((BigInteger*)vb)->n[m] + 48);

  dst[i] = '\0';

  //as we're working on the pointer, check it and restore values if needed
  if (sig == 1)
    ((BigInteger*)vb)->n[((BigInteger*)vb)->count] *= -1;
}

/*
 * equals. Compares two numbers
 *
 * Return 0 if a = b
 *        1 if a > b
 *        2 if a < b
 */
void equals(void* va, void* vb, int* ret) {
  //validate data before treating
  if (validate == 1) {
    validateBI(va);
    validateBI(vb);
  }

  //delegate on static function
  hardEquals(va, vb, ret);
}

/*
 * iniStr. Allocates memory for a BigInteger string, to be used on toString function
 */
void iniStr(char** dst) {
  *dst = (char*)malloc(sizeof(char) * MAX_LENGTH);
}

/*
 * init. Starts BigInteger engine.
 */
void init(void** m) {
  //add
  ((memory*)m)->vt = malloc(sizeof(BigInteger));

  //subtract
  ((memory*)m)->stmp = malloc(sizeof(BigInteger));

  //multiplication
  ((memory*)m)->mpart = malloc(sizeof(BigInteger));
  ((memory*)m)->mret = malloc(sizeof(BigInteger));
  ((memory*)m)->mzero = malloc(sizeof(BigInteger));
  ((memory*)m)->mone = malloc(sizeof(BigInteger));
  ((memory*)m)->mtmp = malloc(sizeof(BigInteger));

  //division
  ((memory*)m)->done = malloc(sizeof(BigInteger));
  ((memory*)m)->dtmp = malloc(sizeof(BigInteger));
  ((memory*)m)->dret = malloc(sizeof(BigInteger));
  ((memory*)m)->dTemp = malloc(sizeof(BigInteger));
  ((memory*)m)->biTemp = malloc(sizeof(BigInteger));

  //root
  ((memory*)m)->sret = malloc(sizeof(BigInteger));
  ((memory*)m)->sraw = malloc(sizeof(BigInteger));
  ((memory*)m)->sbase = malloc(sizeof(BigInteger));
  ((memory*)m)->szero = malloc(sizeof(BigInteger));

  //power
  ((memory*)m)->bres = malloc(sizeof(BigInteger));
  ((memory*)m)->btmp = malloc(sizeof(BigInteger));

  //append
  ((memory*)m)->aaux = malloc(sizeof(BigInteger));

  //BIT
  ((memory*)m)->biBIT = malloc(sizeof(BIT));

  //common values
  _BI_initialize();

  BImemcpy(((memory*)m)->vt, 2);
}

/*
 * getMemorySize. Returns memory size
 */
size_t getMemorySize() {
  return sizeof(memory);
}

/*
 * clean. Cleans a BigInteger, moving 0 as value
 */
void clean(void* va) {
  BImemcpy(va, 0);
}

/*
 * showError. Show or store an error on behalf of the given code.
 * 5x values are CUDA only.
 */
#if CUDA_ENABLED == 1
__host__ __device__
#endif
 static void showError(int k) {
#if BI_SERVICE == 1
  BIReturnCode = k;
#else
#ifdef  __CUDA_ARCH__
  cBIReturnCode = k;
#else
  if (k == 1)
    printf("Error. Limite alcanzado");
  else if (k == 2)
    printf("Error. Datos erróneos en newBI");
  else if (k == 3)
    printf("Error. Puntero erróneo en mul");
  else if (k == 4)
    printf("Error. Puntero erróneo en dvs");
  else if (k == 5)
    printf("Error. Puntero erróneo en divide");
  else if (k == 6)
    printf("Error. Puntero erróneo en sBipow");
  else if (k == 7)
    printf("Error. Puntero erróneo en pAppend");
  else if (k == 8)
    printf("Error. Exponente demasiado grande");
  else if (k == 9)
    printf("Error. Puntero erróneo en nqrt");
  else if (k == 10)
    printf("Error. Puntero erróneo en sub");
  else if (k == 11)
    printf("Error. Puntero erróneo en add");
  else if (k == 98)
    printf("Error. Puntero erróneo en validateBI");
  else if (k == 99)
    printf("Error. Error de validación de datos");
  else
    printf("Error. Error desconocido");

  printf("\n");

  exit(k * -1);
#endif /* __CUDA_ARCH__ (Else) */
#endif /* BI_SERVICE (Else) */
}
#endif /* BI_STANDALONE */
